#include "hip/hip_runtime.h"
//
//  main.cpp
//  RAPIDS
//
//  Created by Aaron on 11/19/18.
//  Copyright © 2018 Aaron Nightingale. All rights reserved.
//
//  This is a simple CPU groupby class (currently only MAX is implemented)
//  one key column and one value column.

#include <iostream>
#include <time.h>
#include <algorithm>
#include "cpuGroupby.h"

#include "groupby.cu"

using namespace std;
int main(int argc, const char * argv[]) {
        clock_t start, end;

        int num_rows = 100000;
        int num_key_cols = 2;
        int num_val_cols = 3;
        int num_distinct_keys = 3;
        if (argc == 2){
                num_rows = atoi(argv[1]);
        }else if(argc ==4){
                num_rows = atoi(argv[1]);
                num_key_cols = atoi(argv[2]);
                num_val_cols = atoi(argv[3]);
        }
        // Setting up the CPU groupby
        cpuGroupby slowGroupby(num_key_cols, num_val_cols, num_rows);

        slowGroupby.fillRand(num_distinct_keys, num_rows);

        int *original_key_columns;
	hipHostMalloc((void**)&original_key_columns, sizeof(int)*num_key_cols*num_rows);
        int *original_value_columns;
	hipHostMalloc((void**)&original_value_columns, sizeof(int)*num_val_cols*num_rows);
        std::copy(slowGroupby.key_columns, slowGroupby.key_columns + num_key_cols*num_rows, original_key_columns);
        std::copy(slowGroupby.value_columns, slowGroupby.value_columns + num_val_cols*num_rows, original_value_columns);
        
        start = clock();

        slowGroupby.groupby();

        end = clock(); 
        float cpu_duration = ((float)end-(float)start)/CLOCKS_PER_SEC; 

        // Insert GPU function calls here...
        int *gpu_output_keys, *gpu_output_values;
        int gpu_output_rows = 0;
        gpu_output_keys = (int *)malloc(slowGroupby.num_key_rows*slowGroupby.num_key_columns * sizeof(int));
        gpu_output_values = (int *)malloc(slowGroupby.num_value_rows*slowGroupby.num_value_columns * sizeof(int));

        start = clock();

        groupby_GPU(original_key_columns, slowGroupby.num_key_columns,
                slowGroupby.num_key_rows, original_value_columns, 
                slowGroupby.num_value_columns, slowGroupby.num_value_rows, 
                slowGroupby.ops, slowGroupby.num_ops,
                gpu_output_keys, gpu_output_values, gpu_output_rows); 
        slowGroupby.printGPUResults(gpu_output_keys, gpu_output_values);

        end = clock(); 
        float gpu_duration = ((float)end-(float)start)/CLOCKS_PER_SEC; 

        cout << "CPU time: " << cpu_duration << "s" << endl;
        cout << "GPU time: " << gpu_duration << "s" << endl;

        slowGroupby.validGPUResult(gpu_output_keys, gpu_output_values, gpu_output_rows);

        hipHostFree(original_value_columns);
        hipHostFree(original_key_columns);
        return 0;
}
