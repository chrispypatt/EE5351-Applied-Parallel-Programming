#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "util.h"
#include "ref_2dhisto.h"
#include "opt_2dhisto.h"

//abandoned, not possible - 1 hour while setting up functional non optimized kernel
// __device__ uint8_t atomicAdd8Int(uint8_t* address, uint8_t val){ //function to write to uint8_t atomically
// 	unsigned long long int* address_as_ull = (unsigned long long int*)address;
// 	unsigned long long int old = *address_as_ull, assumed;
// 	do{
// 		assumed = old;	// READ
// 		old = atomicCAS(address_as_ull, assumed,val + assumed);	// MODIFY + WRITE
// 	} while (assumed != old);
// 	return old;
// }

__global__ void histogram_kernel(uint32_t *buff, long size, int *histo){
	//start index into buffer
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	//stride is however many threads we have running 
	int stride = blockDim.x * gridDim.x; //memory coalescing

	__shared__ unsigned int histo_private[HISTO_WIDTH];
	if (threadIdx.x < HISTO_WIDTH) histo_private[threadIdx.x] = 0;
	__syncthreads();
	
	while(index < size){
		if (histo_private[buff[index]] < UINT8_MAXIMUM){ //Dont rollover!!
			atomicAdd(&(histo_private[buff[index]]), 1);
			index += stride;
		}
	}
	__syncthreads();
	if(threadIdx.x < HISTO_WIDTH){
		atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);
	}
}

__global__ void convert_int2uint8(int *int_histo, int size, uint8_t *histo){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < size){
		if (int_histo[index]>UINT8_MAXIMUM){
			histo[index] = UINT8_MAXIMUM;
		}else{
			histo[index] = (uint8_t)int_histo[index];
		}
	}
}

void opt_2dhisto(uint32_t *input[], size_t height, size_t width, uint8_t bins[HISTO_HEIGHT*HISTO_WIDTH])
{
	long input_size = height*width;
	int histo_size = HISTO_HEIGHT*HISTO_WIDTH;
	//init bins to zero
	memset(bins, 0, HISTO_HEIGHT*HISTO_WIDTH*sizeof(bins[0]));
	//setup our device histogram
	uint32_t* d_input;
	hipMalloc((void**)&d_input, input_size*sizeof(uint32_t));
	for(int i = 0; i < height; ++i){
		//copy each row of input array to device 
		CopyInputToDeviceArray((d_input+i*width), width, input[i]);
    }	

	//uint8_t histogram for output
	uint8_t* d_histo;
	hipMalloc((void**)&d_histo, HISTO_HEIGHT*HISTO_WIDTH*sizeof(uint8_t));
	hipMemset(d_histo, 0, HISTO_HEIGHT*HISTO_WIDTH*sizeof(uint8_t));

	//int histogram for calculation. atomicadd works with this
	int* d_int_histo;
	hipMalloc((void**)&d_int_histo, HISTO_HEIGHT*HISTO_WIDTH*sizeof(int));
	hipMemset(d_int_histo, 0, HISTO_HEIGHT*HISTO_WIDTH*sizeof(int));


	dim3 dimGrid(ceil((float)input_size/(float)BLOCK_SIZE),1,1);
	dim3 dimBlock(BLOCK_SIZE,1,1);

	histogram_kernel<<<dimGrid,dimBlock>>>(d_input,input_size,d_int_histo);
    hipDeviceSynchronize();

	dimGrid.x = (ceil((float)histo_size/(float)BLOCK_SIZE),1,1);

	convert_int2uint8<<<dimGrid,dimBlock>>>(d_int_histo,histo_size,d_histo);
    hipDeviceSynchronize();

	CopyBinsFromDeviceArray(bins,HISTO_HEIGHT,HISTO_WIDTH,d_histo);

	hipFree(d_histo);
	hipFree(d_int_histo);
	hipFree(d_input);
}

/* Include below the implementation of any other functions you need */

//copy data (input) from the host to our device
void CopyInputToDeviceArray(uint32_t* Adevice, size_t width, uint32_t* Ahost){
    int size = width * sizeof(uint32_t);
    hipMemcpy(Adevice, Ahost, size, hipMemcpyHostToDevice);
}

//copy data back from device to our host
void CopyBinsFromDeviceArray( uint8_t* Ahost, size_t height, size_t width,uint8_t* Adevice){
    int size = width * height * sizeof(uint8_t);
    hipMemcpy(Ahost, Adevice, size, hipMemcpyDeviceToHost);
}
