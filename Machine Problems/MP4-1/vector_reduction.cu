#include "hip/hip_runtime.h"
#ifdef _WIN32
#  define NOMINMAX 
#endif

#define NUM_ELEMENTS 512//10000000
#define BLOCK_SIZE 512

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, kernels
#include "vector_reduction_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

int ReadFile(unsigned int*, char* file_name);
unsigned int computeOnDevice(unsigned int* h_data, int array_mem_size);
void copyFromDeviceArray(unsigned int* h_data, const unsigned int*  d_data, int num_elements);
void copyToDeviceArray(unsigned int* d_data, const unsigned int*  h_data, int num_elements);
unsigned int* allocateDeviceArray(unsigned int* data, int num_elements);

extern "C" 
void computeGold( unsigned int* reference, unsigned int* idata, const unsigned int len);


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int num_elements = NUM_ELEMENTS;
    int errorM = 0;

    const unsigned int array_mem_size = sizeof( unsigned int) * num_elements;

    // allocate host memory to store the input data
    unsigned int* h_data = (unsigned int*) malloc( array_mem_size);

    // * No arguments: Randomly generate input data and compare against the 
    //   host's result.
    // * One argument: Read the input data array from the given file.
    switch(argc-1)
    {      
        case 1:  // One Argument
            errorM = ReadFile(h_data, argv[1]);
            if(errorM != num_elements)
            {
                printf("Error reading input file!\n");
                exit(1);
            }
        break;
        
        default:  // No Arguments or one argument
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
                //h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
		h_data[i] = rand()%1000;
            }
        break;  
    }
    // compute reference solution
    unsigned int reference = 0;  
    computeGold(&reference , h_data, num_elements);
    
    // **===-------- Modify the body of this function -----------===**
    unsigned int result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    unsigned int epsilon = 0;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    //printf( "device: %f  host: %f\n", result, reference);
    printf( "device: %d  host: %d\n", result, reference);
    // cleanup memory
    free( h_data);
}

// Read a vector into M (already allocated) from file
int ReadFile(unsigned int* V, char* file_name)
{
    unsigned int data_read = NUM_ELEMENTS;
    FILE* input = fopen(file_name, "r");
    unsigned i = 0;
    for (i = 0; i < data_read; i++) 
        fscanf(input, "%d", &(V[i]));
    return data_read;
}

// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copies it to device, setup grid and thread 
// dimentions, excutes kernel function, and copy result of reduction back
// to h_data.
// Note: unsigned int* h_data is both the input and the output of this function.
unsigned int computeOnDevice(unsigned int* h_data, int num_elements)
{
	int curr_count = num_elements;

	//init data on device
	unsigned int* d_data = allocateDeviceArray(h_data, num_elements);
	copyToDeviceArray(d_data,h_data,num_elements);

	//set kernel size
	dim3 dimGrid(ceil(double(curr_count)/double(2*BLOCK_SIZE)),1,1);
	dim3 dimBlock(BLOCK_SIZE,1,1);

	/* Each loop # elements = # blocks in previous loop.
		Don't need to reallocate memory for each iteration, we just use less. 
		May be smart to give up memory we aren't using anymore */
	while(curr_count > 1){
		reduction<<<dimGrid,dimBlock>>>(d_data,curr_count);
		curr_count = dimGrid.x;
		dimGrid.x = ceil(double(curr_count)/double(2*BLOCK_SIZE));
	}

	copyFromDeviceArray(h_data,d_data,1);//just copying back 1 element
	hipFree(&d_data);
	return h_data[0]; //Input array now reduced to sum of all elements
}

/*------Helper functions to keep computeOnDevice() look clean-------*/

// Allocate a device array of same size as data.
unsigned int* allocateDeviceArray(unsigned int* data, int num_elements){
	int size = num_elements * sizeof(unsigned int);
	unsigned int* d_data = data;
	hipError_t cuda_ret = hipMalloc((void**) &d_data, size);
	if(cuda_ret != hipSuccess) {
		printf("Unable to allocate device memory");
		exit(0);
	}
	return d_data;
}

// Copy a host array to a device array.
void copyToDeviceArray(unsigned int* d_data, const unsigned int*  h_data, int num_elements)
{
    int size = num_elements * sizeof(unsigned int);
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
}

// Copy a device array to a host array.
void copyFromDeviceArray(unsigned int* h_data, const unsigned int*  d_data, int num_elements)
{
    int size = num_elements * sizeof(unsigned int);
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

}
