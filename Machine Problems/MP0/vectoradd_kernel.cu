#include "hip/hip_runtime.h"
/* Vector Addition: C = A + B.
 * Device code.
 */

#ifndef _VECTORADD_KERNEL_H_
#define _VECTORADD_KERNEL_H_

#include <stdio.h>
#include "vectoradd.h"

// Vector addition kernel thread specification
__global__ void VectorAddKernel(Vector A, Vector B, Vector C)
{
	//Add the two vectors
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < VSIZE) {
       C.elements[i] = A.elements[i] + B.elements[i];
    } 
}

#endif // #ifndef _VECTORADD_KERNEL_H_
